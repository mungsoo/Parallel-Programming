#include "hip/hip_runtime.h"
#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


__device__ int blockCounter = 0;

__global__ void scan(float *input, float *output, int len, float *flag, volatile float *preSum) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host
  
  __shared__ int bid;
  __shared__ float array[2 * BLOCK_SIZE];
  __shared__ float preBlockSum;
  int tid = threadIdx.x;
  
  // Dynamically get blockIdx.x
  if(tid == 0)
    bid = atomicAdd(&blockCounter, 1);
  __syncthreads();
  
  // Load the input into shared memory
  int start = bid * BLOCK_SIZE * 2 + tid;
  if(start < len)
    array[tid] = input[start];
  else
    array[tid] = 0;
  if(start + BLOCK_SIZE < len)
    array[tid + BLOCK_SIZE] = input[start + BLOCK_SIZE];
  else
    array[tid + BLOCK_SIZE] = 0;
  
  // Reduction phase
  int stride = 1;
  while(stride < 2 * BLOCK_SIZE)
  {
    __syncthreads();
    int index = (tid + 1) * stride * 2 - 1;
    if(index < 2 * BLOCK_SIZE)
      array[index] += array[index - stride];
    stride *= 2;
  }
  
  // Post scan phase
  stride = BLOCK_SIZE / 2;
  while(stride > 0)
  {
    __syncthreads();
    int index = (tid + 1) * 2 * stride - 1;
    if(index + stride < 2 * BLOCK_SIZE)
      array[index + stride] += array[index];
    stride /= 2;
  }
  
  // Check flag to find if preSum has been writes into memory
  if(tid == 0)
  {
    while(atomicAdd(&flag[bid], 0) == 0);
    preBlockSum = preSum[bid];
    preSum[bid + 1] = array[2 * BLOCK_SIZE - 1] + preBlockSum;
    // use threadfence() to gurantee all the memory writes before it can be seen by all threads =
    __threadfence();
    atomicAdd(&flag[bid + 1], 1);
  }   
  __syncthreads();
  
  array[tid] += preBlockSum;
  array[tid + BLOCK_SIZE] += preBlockSum;
  if(start < len)
    output[start] = array[tid];
  if(start + BLOCK_SIZE < len)
    output[start + BLOCK_SIZE] = array[tid + BLOCK_SIZE];
  
    
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list  
  float *flag;
  volatile float *preSum;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  
  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);
  
  
  int numBlocks = ceil(1.0 * numElements / 2 / BLOCK_SIZE);
  
  
  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&flag, numBlocks * sizeof(float)));
  wbCheck(hipMalloc((void **)&preSum, numBlocks * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  
  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbCheck(hipMemset(flag, 0, numBlocks * sizeof(float)));
  wbCheck(hipMemset(flag, 0xff, sizeof(float)));
  wbCheck(hipMemset((void *)preSum, 0, numBlocks * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  
  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  
  
  //@@ Initialize the grid and block dimensions here
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize(numBlocks, 1, 1);
  
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  
  scan<<<gridSize, blockSize>>>(deviceInput, deviceOutput, numElements, flag, preSum);
  hipDeviceSynchronize();
  
  wbTime_stop(Compute, "Performing CUDA computation");
  
  
  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");
  
  
  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipFree((void *)preSum);
  hipFree(flag);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}